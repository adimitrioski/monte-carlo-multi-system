#include "hip/hip_runtime.h"
#include "../../histogram.cuh"
#include "../../monte_carlo.cuh"
#include "cpu_monte_carlo.cuh"
#include <tbb/tick_count.h>
#include <math.h>
#include <vector>
#include <thrust/extrema.h>
#include <thrust/reduce.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/system/tbb/execution_policy.h>


inline float calculate_mean(
		std::vector<float> &h_vector,
		unsigned long long int num_iterations) {

	float sum = thrust::reduce(thrust::tbb::par, h_vector.begin(), h_vector.end());
	return sum / num_iterations;
};

inline float get_min(
		std::vector<float> &h_ending_values) {

	std::vector<float>::iterator min_iterator = thrust::min_element(thrust::tbb::par, h_ending_values.begin(), h_ending_values.end());
	return *min_iterator;
}

inline float get_max(
		std::vector<float> &h_ending_values) {

	std::vector<float>::iterator max_iterator = thrust::max_element(thrust::tbb::par, h_ending_values.begin(), h_ending_values.end());
	return *max_iterator;
};

inline float calculate_standard_deviation(
		std::vector<float> &h_ending_values,
		float mean,
		unsigned long long int num_iterations) {

	float sum = thrust::reduce(
			thrust::tbb::par,
			thrust::make_transform_iterator(h_ending_values.begin(), SubtractMeanAndSquareFromDataFunctor(mean)),
			thrust::make_transform_iterator(h_ending_values.end(), SubtractMeanAndSquareFromDataFunctor(mean)));

	return sqrt(sum / num_iterations);
};

MonteCarloResult cpu_tbb_run_monte_carlo_simulation(
		MonteCarloRequest monte_carlo_request) {

	std::vector<float> h_ending_values(monte_carlo_request.num_iterations);

	tbb::tick_count start = tbb::tick_count::now();

	thrust::transform(
			thrust::tbb::par,
			thrust::counting_iterator<unsigned long long int>(0),
			thrust::counting_iterator<unsigned long long int>(monte_carlo_request.num_iterations),
			h_ending_values.begin(),
			HostMonteCarloFunctor(
					monte_carlo_request.expected_return,
					monte_carlo_request.volatility,
					monte_carlo_request.time_horizon,
					monte_carlo_request.starting_value,
					monte_carlo_request.annual_investment));

	float mean = calculate_mean(h_ending_values, monte_carlo_request.num_iterations);
	float min = get_min(h_ending_values);
	float max = get_max(h_ending_values);
	float standard_deviation = calculate_standard_deviation(h_ending_values, mean, monte_carlo_request.num_iterations);

	std::vector<float> h_histogram_values;
	std::vector<unsigned int> h_histogram_counts;
	cpu_tbb_sparse_histogram(h_ending_values, h_histogram_values, h_histogram_counts);

	tbb::tick_count end = tbb::tick_count::now();

	float simulation_time = (end - start).seconds() * 1000;

	return fillMonteCarloResult(
			h_histogram_values,
			h_histogram_counts,
			mean,
			min,
			max,
			standard_deviation,
			monte_carlo_request.num_iterations,
			monte_carlo_request.time_horizon,
			simulation_time);
};
